#include "hip/hip_runtime.h"
extern "C" __global__ void computeOneDimComForce(const real4* __restrict__ posq, int nAtoms, float k, float r0,
                                      const int* __restrict__ indices, const float* __restrict__ weights,
                                      unsigned long long* __restrict__ forceBuffer, real* __restrict__ energyBuffer) {
    extern __shared__ float accumulator[];

    // our index
    // this kernel is only run with a single thread block
    int threadIndex = threadIdx.x;

    // zero out the accumulator
    accumulator[threadIndex] = 0.0;

    // each thread adds it's values to the accumulator
    for (int index=threadIndex; index<nAtoms; index+=blockDim.x) {
        // we subtract so that the sign is positive when group2 is to the
        // right of group 1
        accumulator[threadIndex] -= posq[indices[index]].x * weights[index];
    }
    __syncthreads();

    // now do a parallel reduction to get the weighted displacement
    for (unsigned int stride=blockDim.x/2; stride>0; stride>>=1) {
        if (threadIndex < stride) {
            accumulator[threadIndex] += accumulator[threadIndex + stride];
        }
        __syncthreads();
    }

    // compute the energy on thread zero
    if (threadIndex == 0) {
        energyBuffer[0] += 0.5 * k * (accumulator[0] - r0) * (accumulator[0] - r0);
    }
    __syncthreads();

    // compute the forces and store in the buffer
    float factor = k * (accumulator[0] - r0);
    for (int index=threadIndex; index<nAtoms; index+=blockDim.x) {
        float force = factor * weights[index];
        atomicAdd(&forceBuffer[indices[index]], static_cast<unsigned long long>((long long)(force*0x100000000)));
    }
}
